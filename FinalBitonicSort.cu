
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <cmath>
using namespace std;

/* 
  arrayInit fill the array with random values 
  it takes as input the array to fill and its lenght  
*/
void arrayInit(int *arr, unsigned long length) {
    unsigned long i;
    for (i = 0; i < length; ++i) {
        arr[i] = (rand() % 100);
    }
}
/* 
  checkArray check if the array after the bitonic sort is properly sorted 
  it takes as input the array to fill and its lenght
  return true if the array is sorted, false otherwise
  */
bool checkArray(int *arr, unsigned long long length) {
    for (unsigned long long i = 1; i < length; ++i) {
        if(arr[i-1]>arr[i]){
            printf("Array not sorted");
            return false;
        }
    }
    return true;
}

/* 
  swap_values use a temp variable in order to swap two values
  it is performed in GPU and it is used by bitonic_sort_kernel
  it takes as input the two values to swap 
  */
__device__ void swap_values(int *a, int *b){
  int temp = *a;
  *a = *b;
  *b = temp;
}

/* 
 bitonic_sort_kernel performs a single pass of sorting in the GPU 
 it takes as input the array on which to operate, the distance between the element to
 swap in case the check is passed and the size of the sequence to sort
*/
__global__ void bitonic_sort_kernel(int *arr, unsigned long long distance, unsigned long long subSequence_size){
  //get the thread id and the check value using the xor operands
  unsigned long long i, xorCheck; 
  i = threadIdx.x + blockDim.x * blockIdx.x;
  
  //get the element in the array to sort
  xorCheck = i^distance;

  //Sort only the element that are distant enough
  if ((xorCheck)>i) {
    
    //if the operator produce 0 we are in the ascending part
    //of the bitonic sequence
    if ((i & subSequence_size)==0) {
      if (arr[i]>arr[xorCheck]) {
        swap_values(&arr[i],&arr[xorCheck]);
      }
    }

    //otherwise we are in the decending part
    else {
      if (arr[i]<arr[xorCheck]) {
        swap_values(&arr[i],&arr[xorCheck]);
      }
    }
  }
}

/* 
  bitonic_sort performs the operations to sort the array 
  it takes as input the array to sort and its lenght
  */
void bitonic_sort(int *arr, unsigned long long array_size){
    int *cuda_arr; // device array
    size_t size = array_size * sizeof(int); // size * 4 byte
    
    //allocate memory on device
    hipMalloc((void**) &cuda_arr, size); 
    //copy the original array to the device one
    hipMemcpy(cuda_arr, arr, size, hipMemcpyHostToDevice); 

    //set the number of threads per blocks and calculate the number of blocks
    int thread_dim=128;
    int block_dim=(array_size + thread_dim - 1) / thread_dim;

    
    unsigned long long distance, subSequence_size; 

    //initialize the time recording without cudamemcpy
    hipEvent_t start, stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //iterate through the array sorting sequence of ascending dimention over distances
    //of decending dimention
    for (subSequence_size = 2; subSequence_size <= array_size; subSequence_size <<= 1) 
      for (distance = subSequence_size >> 1; distance > 0; distance = distance >> 1)        
        bitonic_sort_kernel<<<block_dim, thread_dim>>>(cuda_arr, distance, subSequence_size);
        
    //calculate elapsed time withouth the cudaMemcpy
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("The elapsedtime in GPU without memcopy was: %.3f ms\n", elapsed);
    
    //copy back to the host array
    hipMemcpy(arr, cuda_arr, size, hipMemcpyDeviceToHost); 
    //free device memory
    hipFree(cuda_arr); 
   
}

int main(int argc, char** argv){
  //counter for the sum of the elapsed time for the numbers of iteration  
  double tot=0;
  double min = 99999999999; // initialization value 
  int power = atoi(argv[1]); 
  //number of elements must be a power of 2 so the dimension is 2^power
  unsigned long long array_size = 1<<power; 
  printf("array_size %llu --> 2^%i\n", array_size, power);
  //number of iterations
  int iter=30; 

  for(int h=0; h<iter; h++){
    //allocate host array
    int *arr = (int*) malloc( array_size * sizeof(int)); 
    //initialize the host array with random values
    arrayInit(arr, array_size); 

    hipEvent_t startExt, outExt;
    //contains the elapsed time on the bitonic sort with bitonic sort
    float elapsed; 
    hipEventCreate(&startExt);
    hipEventCreate(&outExt);
    hipEventRecord(startExt, 0);
    //=========================================//
    bitonic_sort(arr, array_size);
    //=========================================//
    hipEventRecord(outExt, 0);
    hipEventSynchronize(outExt);
    hipEventElapsedTime(&elapsed, startExt, outExt);
    hipEventDestroy(startExt);
    hipEventDestroy(outExt);
    printf("The elapsedtime in GPU with memcopy was: %.3f ms\n", elapsed);

    //check if sort has been done correctly
    if(checkArray(arr,array_size)){ 
        printf("Array sorted correctly\n");
    }
    //free host memory
    free(arr); 
    //add the current iteration to the total
    tot +=elapsed; 
    
    //calculate the minimum over the various iteration
    if(elapsed<min){
      min = elapsed;
    }
  }
  printf("Average elapsed time: %.3f\n",tot/iter);
  printf("Minimum time: %.3f\n",min);
}